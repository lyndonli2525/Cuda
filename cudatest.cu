#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>

__global__ void sum_arr_on_host(float *A, float *B, float *C, const int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
        for (int i = index; i < N; i += stride)
            C[i] = A[i] + B[i];
}

void init_data(float *arr, int size) {
  time_t t;
  srand((unsigned int)time(&t));
  for (int i = 0; i < size; i++) {
    arr[i] = (float)(rand() & 0xFF) / 10.0f;
  }
}

int main(int argc, char **argv) {
    int num_elems = 16000000;
    int num_bytes = num_elems * sizeof(float);
    float *A, *B, *C;
    int blockSize = 256;
    int numBlocks = (num_elems + blockSize -1) / blockSize;
    hipMallocManaged(&A, num_bytes);
    hipMallocManaged(&B, num_bytes);
    hipMallocManaged(&C, num_bytes);
    init_data(A, num_elems);
    init_data(B, num_elems);
    init_data(C, num_elems);
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(A, num_bytes, device, NULL);
    hipMemPrefetchAsync(B, num_bytes, device, NULL);
    sum_arr_on_host<<<numBlocks, blockSize>>>(A, B, C, num_elems);
    hipFree(A);
    hipFree(B);
    hipFree(C);
    return 0;
}